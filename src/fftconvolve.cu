/**
    fftconvolve.cu
    Compute complex-complex and real-real FFT convolutions on the GPU

    @author: Konstantinos Iliakis (konstantinos.iliakis@cern.ch)
    @version: 1.0 17/10/2017
*/


#include "cuda_utils.h"
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <thrust/device_vector.h>

using namespace std;

extern "C" {
    std::unordered_map<std::string, std::vector<double>> timerMap;

    void reset_timer()
    {
        timerMap.clear();
    }

    void report_timer()
    {
        double total_time = 0;
        if (timerMap.find("total_time") != timerMap.end())
            total_time = std::accumulate(timerMap["total_time"].begin() + 1,
                                         timerMap["total_time"].end(), 0.0);
        for (auto &kv : timerMap) {
            auto v = kv.second;
            double sum = std::accumulate(v.begin() + 1, v.end(), 0.0);
            printf("[%s] Calls: %d, Average time: %.3lf ms, Global Percentage: %.2lf %\n",
                   kv.first.c_str(), v.size(), sum / v.size(), 100.0 * sum / total_time);
        }
    }



    struct complexMultiplier
    {
        double scale;
        complexMultiplier(double scale): scale(scale) {};

        __host__ __device__
        hipfftDoubleComplex operator() (const hipfftDoubleComplex &v1,
                                       const hipfftDoubleComplex &v2) const
        {
            hipfftDoubleComplex res;
            res.x = (v1.x * v2.x - v1.y * v2.y) * scale;
            res.y = (v1.x * v2.y + v1.y * v2.x) * scale;
            return res;
        }
    };

    /**
        Creates an FFT Plan if it has not been yet initialized

        @plan: Pointer to the plan that will be created/initialized
        @size: Size of the FFT for which this plan will be used
        @type: Type of the FFT
        @batch: Number of FFTs of the specified size that will be computed together.

    */
    void create_plan(hipfftHandle *plan, size_t size, hipfftType type, int batch = 1)
    {
        size_t workSize;
        int ret = hipfftGetSize(*plan, &workSize);
        if (ret == HIPFFT_INVALID_PLAN) {
            if (hipfftPlan1d(plan, size, type, batch) != HIPFFT_SUCCESS) {
                fprintf(stderr, "CUFFT error: Plan creation failed");
            }
        }
    }


    /**
        Computes the FFT convolution of two complex signals

        @signal: The first signal. This is a pointer to host(CPU) memory
        @signalSize: The signal size
        @kernel: The second signal, also called kernel. This is a pointer to
                 host(CPU) memory
        @kernelSize: The kernel size
        @result: Pointer to host(CPU) memory that contains the convolution result.
                 Sufficient memory ((singalSize + kernelSize -1) * sizeof(hipfftDoubleComplex))
                 has to be allocated before calling the function.
        @d_in: Pointer to GPU memory used by the function. The size of the memory region
                has to be at least 2 * (signalSize + kernelSize - 1)
        @fwplan: An integer handle used to store the forward FFT plan.
        @bwplan: An integer handle used to store the backward FFT plan.
    */
    void convolve_complex(hipfftDoubleComplex * signal, int signalSize,
                          hipfftDoubleComplex * kernel, int kernelSize,
                          hipfftDoubleComplex * result,
                          hipfftDoubleComplex * d_in,
                          hipfftHandle *fwplan,
                          hipfftHandle *bwplan)
    {


        // timer timer, globalTimer;
        // globalTimer.restart();
        size_t real_size = signalSize + kernelSize - 1;

        // timer.restart();
        hipMemset(d_in, 0, 2 * real_size * sizeof(hipfftDoubleComplex));
        // timerMap["memset"].push_back(timer.elapsed());

        // timer.restart();
        hipMemcpy(d_in, signal, signalSize * sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice);
        hipMemcpy(d_in + real_size, kernel, kernelSize * sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice);
        // timerMap["memcpy"].push_back(timer.elapsed());

        // timer.restart();
        create_plan(fwplan, real_size, HIPFFT_Z2Z, 2);
        create_plan(bwplan, real_size, HIPFFT_Z2Z, 1);
        // timerMap["create_plans"].push_back(timer.elapsed());

        // timer.restart();
        hipfftExecZ2Z(*fwplan, d_in, d_in, HIPFFT_FORWARD);
        // timerMap["forward"].push_back(timer.elapsed());

        // timer.restart();
        thrust::device_ptr<hipfftDoubleComplex> a(d_in);
        thrust::transform(a, a + real_size, a + real_size, a,
                          complexMultiplier(1.0 / real_size));
        // timerMap["multiply"].push_back(timer.elapsed());

        // timer.restart();
        hipfftExecZ2Z(*bwplan, d_in, d_in, HIPFFT_BACKWARD);
        // timerMap["backward"].push_back(timer.elapsed());

        // timer.restart();
        hipMemcpy(result, d_in, real_size * sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);
        // timerMap["copy_back"].push_back(timer.elapsed());
        // timerMap["total_time"].push_back(globalTimer.elapsed());
    }

    /**
        Computes the FFT convolution of two real signals

        @signal: The first signal. This is a pointer to host(CPU) memory
        @signalSize: The signal size
        @kernel: The second signal, also called kernel. This is a pointer to
                 host(CPU) memory
        @kernelSize: The kernel size
        @result: Pointer to host(CPU) memory where the convolution result will be copied.
                 Sufficient memory ((signalSize + kernelSize - 1)*sizeof(double))
                 has to be allocated before calling the function.
        @fwplan: An integer handle used to store the forward FFT plan.
        @bwplan: An integer handle used to store the backward FFT plan.
    */
    void convolve_real(double * signal, int signalSize,
                       double * kernel, int kernelSize,
                       double * result,
                       hipfftHandle *fwplan,
                       hipfftHandle *bwplan)
    {
        hipfftDoubleComplex *d_out;
        double *d_in;

        size_t real_size = signalSize + kernelSize - 1;
        size_t complex_size = real_size / 2 + 1;

        hipMalloc((void**)&d_in, 2 * real_size * sizeof(double));
        hipMalloc((void**)&d_out, 2 * complex_size * sizeof(hipfftDoubleComplex));

        hipMemset(d_in, 0, 2 * real_size * sizeof(double));
        hipMemcpy(d_in, signal, signalSize * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_in + real_size, kernel, kernelSize * sizeof(double), hipMemcpyHostToDevice);

        create_plan(fwplan, real_size, HIPFFT_D2Z, 2);
        create_plan(bwplan, real_size, HIPFFT_Z2D);

        hipfftExecD2Z(*fwplan, d_in, d_out);

        thrust::device_ptr<hipfftDoubleComplex> a(d_out);
        thrust::transform(a, a + complex_size, a + complex_size, a,
                          complexMultiplier(1.0 / real_size));

        hipfftExecZ2D(*bwplan, d_out, d_in);

        hipMemcpy(result, d_in, real_size * sizeof(double), hipMemcpyDeviceToHost);

        hipFree(d_in);
        hipFree(d_out);
    }


    /**
        Computes the FFT convolution of two real signals

        @signal: The first signal. This is a pointer to GPU memory
        @signalSize: The signal size
        @kernel: The second signal, also called kernel. This is a pointer to
                 GPU memory
        @kernelSize: The kernel size
        @result: Pointer to GPU memory where the convolution result will be copied.
                 Sufficient memory ((signalSize + kernelSize - 1)*sizeof(double))
                 has to be allocated before calling the function.
        @fwplan: An integer handle used to store the forward FFT plan.
        @bwplan: An integer handle used to store the backward FFT plan.
    */
    void convolve_real_no_memcpy(double * signal, int signalSize,
                                 double * kernel, int kernelSize,
                                 double * result,
                                 hipfftHandle *fwplan,
                                 hipfftHandle *bwplan)
    {
        hipfftDoubleComplex *d_out;
        size_t real_size = signalSize + kernelSize - 1;
        size_t complex_size = real_size % 2 == 1 ?
                              ((real_size + 1) / 2) : (real_size / 2 + 1);

        hipMalloc((void**)&d_out, 2 * complex_size * sizeof(hipfftDoubleComplex));

        create_plan(fwplan, real_size, HIPFFT_D2Z);
        create_plan(bwplan, real_size, HIPFFT_Z2D);

        hipfftExecD2Z(*fwplan, signal, d_out);
        hipfftExecD2Z(*fwplan, kernel, d_out + complex_size);

        thrust::device_ptr<hipfftDoubleComplex> a(d_out);
        thrust::transform(a, a + complex_size, a + complex_size, a,
                          complexMultiplier(1.0 / real_size));

        hipfftExecZ2D(*bwplan, d_out, result);

        hipFree(d_out);
    }

}
